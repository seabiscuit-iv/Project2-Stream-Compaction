#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <vector>

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void Reduce(int n, int* data, int k) {
            CALCULATE_TID_AUTO;

            if ( tid >= n || (tid + 1) % (2 * k) != 0 ) {
                return;
            }

            data[tid] = data[tid] + data[tid - k];
        }

        __global__ void DownSweep(int n, int* data, int k) {
            CALCULATE_TID_AUTO;
            tid = 2*tid + 1;

            k = n / (2 << k);

            if ( tid >= n || (tid + 1) % (2 * k) != 0 ) {
                return;
            }

            int temp = data[tid];
            data[tid] = data[tid] + data[tid - k];
            data[tid - k] = temp;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata, bool timer_enabled) {
            if (timer_enabled) timer().startGpuTimer();

            int og_n = n;
            n = 1 << ilog2ceil(n);
            
            int* data;
            hipMalloc((void**)&data, n * sizeof(int));
            hipMemset(data, 0, n * sizeof(int));
            hipMemcpy(data, idata, og_n * sizeof(int), hipMemcpyHostToDevice);

            int num_iterations = ilog2ceil(n);

            CALCULATE_BLOCK_THREAD_SIZE_AUTO(n, BLOCK_SIZE);

            // up sweep
            for(int i = 0; i < num_iterations; i++) {
                Reduce<<<blocksPerGrid, threadsPerBlock>>>(n, data, 1 << i);
            }

            // down sweep
            hipMemset(data + (n-1), 0, sizeof(int));
            CALCULATE_BLOCK_THREAD_SIZE(n/2, BLOCK_SIZE, blocksPerGrid_new, threadsPerBlock_new);
            for (int k = 0; k < num_iterations; k++) {
                DownSweep<<<blocksPerGrid_new, threadsPerBlock_new>>>(n, data, k);
            }

            hipMemcpy(odata, data, og_n * sizeof(int), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();

            hipFree(data);

            if (timer_enabled) timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            int *read, *flags, *scanout, *write;
            hipMalloc((void**)&read, n * sizeof(int));
            hipMalloc((void**)&flags, n * sizeof(int));
            hipMalloc((void**)&scanout, n * sizeof(int));

            hipMemcpy(read, idata, n * sizeof(int), hipMemcpyHostToDevice);

            CALCULATE_BLOCK_THREAD_SIZE_AUTO(n, BLOCK_SIZE);

            StreamCompaction::Common::kernMapToBoolean<<<blocksPerGrid, threadsPerBlock>>>(n, flags, read);

            scan(n, scanout, flags, false);

            int scanout_end = 0;
            int flags_end = 0;
            hipMemcpy(&scanout_end, scanout + (n-1), sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&flags_end, flags + (n-1), sizeof(int), hipMemcpyDeviceToHost);

            int len = scanout_end + flags_end;
            
            hipMalloc((void**)&write, len * sizeof(int));

            StreamCompaction::Common::kernScatter<<<blocksPerGrid, threadsPerBlock>>>(n, write, read, flags, scanout);

            hipMemcpy(odata, write, len * sizeof(int), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();

            hipFree(read);
            hipFree(flags);
            hipFree(scanout);
            hipFree(write);

            timer().endGpuTimer();

            return len;
        }
    }
}
