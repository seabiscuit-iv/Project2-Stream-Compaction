#include "hip/hip_runtime.h"
#include "thread_efficient.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include <vector>

namespace StreamCompaction {
    namespace ThreadEfficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


        __global__ void Reduce(int n, int thread_mult, int* data, int k) {
            CALCULATE_TID_AUTO;
            tid = (tid * thread_mult) + (thread_mult - 1);

            if ( tid >= n || tid < k) {
                return;
            }

            data[tid] = data[tid] + data[tid - k];
        }

        __global__ void DownSweep(int n, int thread_mult, int* data, int k) {
            CALCULATE_TID_AUTO;
            tid = (tid * thread_mult) + (thread_mult - 1);
            tid = 2*tid + 1;

            if ( tid >= n || tid < k) {
                return;
            }

            int temp = data[tid];
            data[tid] = data[tid] + data[tid - k];
            data[tid - k] = temp;
        }

        void scan(int n, int *odata, const int *idata, bool timer_enabled) {
            int og_n = n;
            n = 1 << ilog2ceil(n);
            
            int* data;
            hipMalloc((void**)&data, n * sizeof(int));
            hipMemset(data, 0, n * sizeof(int));
            hipMemcpy(data, idata, og_n * sizeof(int), hipMemcpyHostToDevice);

            if (timer_enabled) timer().startGpuTimer();

            int num_iterations = ilog2ceil(n);

            // up sweep
            for(int i = 0; i < num_iterations; i++) {
                int thread_mult = 1 << (i+1);
                // int thread_mult = 1;

                CALCULATE_BLOCK_THREAD_SIZE_AUTO((n + thread_mult - 1) / thread_mult, BLOCK_SIZE);
                // printf("KERNEL: %d, %d\n", blocksPerGrid, threadsPerBlock);
                Reduce<<<blocksPerGrid, threadsPerBlock>>>(n, thread_mult, data, 1 << i);
            }

            num_iterations++;

            // down sweep
            hipMemset(data + (n-1), 0, sizeof(int));
            for (int k = 0; k < num_iterations; k++) {
                int thread_mult = (n / (1 << k));
                // int thread_mult = 1;

                CALCULATE_BLOCK_THREAD_SIZE((n + thread_mult - 1) / (thread_mult), BLOCK_SIZE, blocksPerGrid_new, threadsPerBlock_new);
                DownSweep<<<blocksPerGrid_new, threadsPerBlock_new>>>(n, thread_mult, data, n / (1 << k));
            }

            if (timer_enabled) timer().endGpuTimer();

            hipMemcpy(odata, data, og_n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(data);
        }   

        int compact(int n, int *odata, const int *idata) {
            int *read, *flags, *scanout, *write;
            hipMalloc((void**)&read, n * sizeof(int));
            hipMalloc((void**)&flags, n * sizeof(int));

            hipMemcpy(read, idata, n * sizeof(int), hipMemcpyHostToDevice);

            // {
                int s_n = n;
                int og_n = s_n;
                s_n = 1 << ilog2ceil(s_n);
                
                hipMalloc((void**)&scanout, s_n * sizeof(int));
                int* data = scanout;
                hipMemset(data, 0, s_n * sizeof(int));
            // }

            timer().startGpuTimer();

            CALCULATE_BLOCK_THREAD_SIZE_AUTO(n, BLOCK_SIZE);
            StreamCompaction::Common::kernMapToBoolean<<<blocksPerGrid, threadsPerBlock>>>(n, flags, read);
            // scan(n, scanout, flags, false);

            // {
                hipMemcpy(data, flags, og_n * sizeof(int), hipMemcpyDeviceToDevice);

                int num_iterations = ilog2ceil(s_n);

                // up sweep
                for(int i = 0; i < num_iterations; i++) {
                    int thread_mult = 1 << (i+1);
                    // int thread_mult = 1;

                    CALCULATE_BLOCK_THREAD_SIZE_AUTO((s_n + thread_mult - 1) / thread_mult, BLOCK_SIZE);
                    // printf("KERNEL: %d, %d\n", blocksPerGrid, threadsPerBlock);
                    Reduce<<<blocksPerGrid, threadsPerBlock>>>(s_n, thread_mult, data, 1 << i);
                }

                num_iterations++;

                // down sweep
                hipMemset(data + (s_n-1), 0, sizeof(int));
                for (int k = 0; k < num_iterations; k++) {
                    int thread_mult = (s_n / (1 << k));
                    // int thread_mult = 1;

                    CALCULATE_BLOCK_THREAD_SIZE((s_n + thread_mult - 1) / (thread_mult), BLOCK_SIZE, blocksPerGrid_new, threadsPerBlock_new);
                    DownSweep<<<blocksPerGrid_new, threadsPerBlock_new>>>(s_n, thread_mult, data, s_n / (1 << k));
                }
            // }
            
            int scanout_end = 0;
            int flags_end = 0;
            hipMemcpy(&scanout_end, scanout + (n-1), sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&flags_end, flags + (n-1), sizeof(int), hipMemcpyDeviceToHost);

            int len = scanout_end + flags_end + 1;
            
            hipMalloc((void**)&write, len * sizeof(int));
            StreamCompaction::Common::kernScatter<<<blocksPerGrid, threadsPerBlock>>>(n, write, read, flags, scanout);
            hipDeviceSynchronize();

            timer().endGpuTimer();

            hipMemcpy(odata, write, len * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(read);
            hipFree(flags);
            hipFree(scanout);
            hipFree(write);

            return len;
        }
    }
}